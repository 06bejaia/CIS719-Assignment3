#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

const int N = 4;
std::vector<int> result(N, 0);

__global__ void cuda_kernel(int* d_matrix, int* d_vec, int* d_result, int n) {
    int i = threadIdx.x;
    if (i < n) {
        for (int j = 0; j < n; j++) {
            d_result[i] += d_matrix[i * n + j] * d_vec[j];
        }
    }
}

int main() {
    int h_matrix[N][N] = {{1, 2, 3, 4}, {5, 6, 7, 8}, {9, 10, 11, 12}, {13, 14, 15, 16}};
    int h_vec[N] = {1, 2, 3, 4};
    int *d_matrix, *d_vec, *d_result;

    hipMalloc(&d_matrix, N * N * sizeof(int));
    hipMalloc(&d_vec, N * sizeof(int));
    hipMalloc(&d_result, N * sizeof(int));

    hipMemcpy(d_matrix, h_matrix, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, N * sizeof(int), hipMemcpyHostToDevice);

    cuda_kernel<<<1, N>>>(d_matrix, d_vec, d_result, N);

    hipMemcpy(result.data(), d_result, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_matrix);
    hipFree(d_vec);
    hipFree(d_result);

    std::cout << "Result: ";
    for (int r : result) std::cout << r << " ";
    std::cout << std::endl;
    return 0;
}
